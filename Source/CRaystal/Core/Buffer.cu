#include <hip/hip_runtime.h>

#include "Buffer.h"
#include "Utils/CudaUtils.h"

namespace CRay {
DeviceBuffer::DeviceBuffer(int sizeInBytes)
    : mpDeviceData(nullptr), mSize(sizeInBytes) {
    CRAYSTAL_ASSERT(sizeInBytes > 0);
    CRAYSTAL_CUDA_CHECK(hipMalloc(&mpDeviceData, sizeInBytes));
}

DeviceBuffer::~DeviceBuffer() {
    free();
    mpDeviceData = nullptr;
}

DeviceBuffer::DeviceBuffer(DeviceBuffer&& other) noexcept
    : mpDeviceData(other.mpDeviceData), mSize(other.mSize) {
    other.mpDeviceData = nullptr;
    other.mSize = 0;
}

DeviceBuffer& DeviceBuffer::operator=(DeviceBuffer&& other) noexcept {
    if (this != &other) {
        if (mpDeviceData) {
            free();
        }
        mpDeviceData = other.mpDeviceData;
        mSize = other.mSize;
        other.mpDeviceData = nullptr;
        other.mSize = 0;
    }
    return *this;
}

void* DeviceBuffer::data() { return mpDeviceData; }

const void* DeviceBuffer::data() const { return mpDeviceData; }

int DeviceBuffer::size() const { return mSize; }

void DeviceBuffer::copyFromHost(const void* pHostData) {
    CRAYSTAL_ASSERT(mpDeviceData != nullptr && pHostData != nullptr);
    CRAYSTAL_ASSERT(mSize != 0);

    CRAYSTAL_CUDA_CHECK(
        hipMemcpy(mpDeviceData, pHostData, mSize, hipMemcpyHostToDevice));
}

void DeviceBuffer::copyToHost(void* pHostData) const {
    CRAYSTAL_ASSERT(mpDeviceData != nullptr && pHostData != nullptr);
    CRAYSTAL_ASSERT(mSize != 0);

    CRAYSTAL_CUDA_CHECK(
        hipMemcpy(pHostData, mpDeviceData, mSize, hipMemcpyDeviceToHost));
}

void DeviceBuffer::memset(unsigned char value) {
    CRAYSTAL_ASSERT(mpDeviceData != nullptr);
    CRAYSTAL_ASSERT(mSize != 0);
    CRAYSTAL_CUDA_CHECK(hipMemset(mpDeviceData, value, mSize));
}

void DeviceBuffer::free() {
    CRAYSTAL_CUDA_CHECK(hipFree(mpDeviceData));
    mpDeviceData = nullptr;
}

}  // namespace CRay
