#include "hip/hip_runtime.h"
#include <algorithm>
#include <numeric>

#include "BVH.h"
#include "Scene/Scene.h"
namespace CRay {

struct StackEntry {
    uint32_t nodeIndex;
    float tMin;
};

CRAYSTAL_DEVICE bool BVHData::intersect(const TriangleMeshSOA& meshSOA,
                                        RayHit& rayHit) const {
    Float _;
    if (!blas.bounds.intersect(rayHit.ray, _)) {
        return false;
    }

    constexpr int MAX_STACK = 64;
    StackEntry stack[MAX_STACK];
    int stackPtr = 0;

    stack[stackPtr++] = {blas.rootIndex, 0.0f};

    bool isHit = false;
    float closestT = rayHit.hitT;

    while (stackPtr > 0) {
        const StackEntry entry = stack[--stackPtr];
        const BVHNode& node = blasNodes[entry.nodeIndex];

        if (entry.tMin > closestT) {
            continue;
        }

        if (std::holds_alternative<BVHNode::LeafProp>(node.props)) {
            const auto& leafProp = std::get<BVHNode::LeafProp>(node.props);
            for (uint32_t i = 0; i < leafProp.leafCount; ++i) {
                PrimitiveID primID = leafProp.leafOffset + i;

                if (intersectShape(primID, meshSOA, rayHit)) {
                    isHit = true;
                    // closestT = rayHit.hitT;
                }
            }
        } else {
            const auto& internalProp =
                std::get<BVHNode::InternalProp>(node.props);
            const BVHNode& leftChild = blasNodes[internalProp.left];
            const BVHNode& rightChild = blasNodes[internalProp.right];

            float leftT = kFltInf;
            float rightT = kFltInf;
            bool hitLeft = leftChild.bounds.intersect(rayHit.ray, leftT);
            bool hitRight = rightChild.bounds.intersect(rayHit.ray, rightT);

            if (hitLeft && hitRight) {
                if (leftT > rightT) {
                    if (leftT < closestT) {
                        stack[stackPtr++] = {internalProp.left, leftT};
                    }
                    if (rightT < closestT) {
                        stack[stackPtr++] = {internalProp.right, rightT};
                    }
                } else {
                    if (rightT < closestT) {
                        stack[stackPtr++] = {internalProp.right, rightT};
                    }
                    if (leftT < closestT) {
                        stack[stackPtr++] = {internalProp.left, leftT};
                    }
                }
            } else if (hitLeft && leftT < closestT) {
                stack[stackPtr++] = {internalProp.left, leftT};
            } else if (hitRight && rightT < closestT) {
                stack[stackPtr++] = {internalProp.right, rightT};
            }
        }
    }

    return isHit;
}

BVH::BVH() {}

void BVH::build(SceneData& data) {
    logInfo("Building BVH");
    // 1. Collect triangle data
    gatherTriangles(data);

    // 2. Prepare data
    std::vector<uint32_t> indices(mBuildTris.size());
    std::iota(indices.begin(), indices.end(), 0);

    mNodes.clear();
    mNodes.reserve(2 * mBuildTris.size());

    // 3. Recursive construction
    AABB rootBounds;
    uint32_t rootIndex =
        buildRecursive(indices, 0, indices.size(), 0, rootBounds);

    // 4. Reorder mesh data
    createFinalBVH(data, indices, rootBounds, rootIndex);

    mNodes.clear();
    mBuildTris.clear();

    logInfo(
        "BVH build stats:\n\t\t\tMax depth: {}\n\t\t\tMax leaf node: "
        "{}\n\t\t\tNode count: {}",
        mBuildStats.maxDepth, mBuildStats.maxLeafNodeCount,
        mBuildStats.nodeCount);
}

BVH::DeviceView BVH::getDeviceView() const { return mView; }

void BVH::gatherTriangles(const SceneData& data) {
    size_t totalTris = 0;
    for (const auto& mesh : data.meshes) {
        totalTris += mesh.index.size() / 3;
    }

    mBuildTris.clear();
    mBuildTris.reserve(totalTris);

    for (size_t meshIdx = 0; meshIdx < data.meshes.size(); ++meshIdx) {
        const auto& mesh = data.meshes[meshIdx];
        for (size_t i = 0; i < mesh.index.size(); i += 3) {
            BuildTriangle tri;
            Float3 v0 = mesh.position[mesh.index[i]];
            Float3 v1 = mesh.position[mesh.index[i + 1]];
            Float3 v2 = mesh.position[mesh.index[i + 2]];

            tri.centroid = (v0 + v1 + v2) / 3.0f;
            tri.bounds |= v0;
            tri.bounds |= v1;
            tri.bounds |= v2;
            tri.meshIndex = meshIdx;
            tri.triIndex = i / 3;

            mBuildTris.push_back(tri);
        }
    }
}

uint32_t BVH::buildRecursive(std::vector<uint32_t>& indices, uint32_t start,
                             uint32_t count, uint32_t depth, AABB& nodeBounds) {
    nodeBounds = AABB();
    for (uint32_t i = start; i < start + count; ++i) {
        nodeBounds |= mBuildTris[indices[i]].bounds;
    }

    BVHNode node;
    node.bounds = nodeBounds;

    // Leaf node
    if (count <= MIN_TRIS_PER_LEAF || depth >= MAX_DEPTH) {
        BVHNode::LeafProp leaf;
        leaf.leafOffset = start;
        leaf.leafCount = count;
        node.props = leaf;
        mNodes.push_back(node);
        mBuildStats.maxLeafNodeCount =
            std::max<uint32_t>(mBuildStats.maxLeafNodeCount, count);
        return mNodes.size() - 1;
    }

    auto split = findBestSplit(indices, start, count, nodeBounds);
    uint32_t splitAxis = split.first;
    float splitPos = split.second;

    // Split triangles
    auto mid = std::partition(
        indices.begin() + start, indices.begin() + start + count,
        [&](uint32_t idx) {
            return mBuildTris[idx].centroid[splitAxis] < splitPos;
        });

    uint32_t leftCount = std::distance(indices.begin() + start, mid);

    // Create leaf if split failed
    if (leftCount == 0 || leftCount == count) {
        BVHNode::LeafProp leaf;
        leaf.leafOffset = start;
        leaf.leafCount = count;
        node.props = leaf;
        mNodes.push_back(node);
        mBuildStats.maxLeafNodeCount =
            std::max<uint32_t>(mBuildStats.maxLeafNodeCount, count);
        return mNodes.size() - 1;
    }

    // Construct children recursively
    AABB leftBounds, rightBounds;
    uint32_t leftChild =
        buildRecursive(indices, start, leftCount, depth + 1, leftBounds);
    uint32_t rightChild = buildRecursive(
        indices, start + leftCount, count - leftCount, depth + 1, rightBounds);

    BVHNode::InternalProp internal;
    internal.left = leftChild;
    internal.right = rightChild;
    node.props = internal;
    mNodes.push_back(node);

    mBuildStats.maxDepth = std::max(mBuildStats.maxDepth, depth);
    return mNodes.size() - 1;
}

std::pair<uint32_t, Float> BVH::findBestSplit(
    const std::vector<uint32_t>& indices, uint32_t start, uint32_t count,
    const AABB& nodeBounds) {
    Float bestCost = kFltInf;
    uint32_t bestAxis = 0;
    Float bestSplit = 0.0f;

    // Split each axis
    for (uint32_t axis = 0; axis < 3; ++axis) {
        // Init bins
        std::array<SAHBin, NUM_BINS> bins;
        Float3 extent = nodeBounds.diagonal();
        Float scale = NUM_BINS / extent[axis];

        // Assign triangle to bins
        for (uint32_t i = start; i < start + count; ++i) {
            const auto& tri = mBuildTris[indices[i]];
            int binIndex = std::min(
                NUM_BINS - 1,
                static_cast<int>((tri.centroid[axis] - nodeBounds.pMin[axis]) *
                                 scale));
            bins[binIndex].bounds |= tri.bounds;
            bins[binIndex].triCount++;
        }

        // Scan best split from left to right
        std::array<AABB, NUM_BINS - 1> leftBounds;
        std::array<AABB, NUM_BINS - 1> rightBounds;
        std::array<int, NUM_BINS - 1> leftCount{0};
        std::array<int, NUM_BINS - 1> rightCount{0};

        // Left accumulate
        AABB currentLeft;
        int currentLeftCount = 0;
        for (int i = 0; i < NUM_BINS - 1; ++i) {
            currentLeft |= bins[i].bounds;
            currentLeftCount += bins[i].triCount;
            leftBounds[i] = currentLeft;
            leftCount[i] = currentLeftCount;
        }

        // Right accumulate
        AABB currentRight;
        int currentRightCount = 0;
        for (int i = NUM_BINS - 1; i > 0; --i) {
            currentRight |= bins[i].bounds;
            currentRightCount += bins[i].triCount;
            rightBounds[i - 1] = currentRight;
            rightCount[i - 1] = currentRightCount;
        }

        // Evaluate possible splits
        for (int i = 0; i < NUM_BINS - 1; ++i) {
            Float splitPos =
                nodeBounds.pMin[axis] + (i + 1) * extent[axis] / NUM_BINS;

            Float cost = evaluateSAH(nodeBounds, leftBounds[i], rightBounds[i],
                                     leftCount[i], rightCount[i]);

            if (cost < bestCost) {
                bestCost = cost;
                bestAxis = axis;
                bestSplit = splitPos;
            }
        }
    }
    return {bestAxis, bestSplit};
}

Float calcAABBSurfaceArea(const AABB& aabb) {
    Float3 xyz = aabb.diagonal();
    return (xyz.x * xyz.y + xyz.x * xyz.z + xyz.y * xyz.z) * 2.0;
}

Float BVH::evaluateSAH(const AABB& nodeBounds, const AABB& leftBounds,
                       const AABB& rightBounds, int leftCount, int rightCount) {
    const float traversalCost = 1.0f;
    const float intersectionCost = 1.0f;

    Float leftSA = calcAABBSurfaceArea(leftBounds);
    Float rightSA = calcAABBSurfaceArea(rightBounds);
    Float rootSA = calcAABBSurfaceArea(nodeBounds);

    return traversalCost + intersectionCost *
                               (leftCount * leftSA + rightCount * rightSA) /
                               rootSA;
}

void BVH::createFinalBVH(SceneData& data,
                         const std::vector<uint32_t>& finalIndices,
                         const AABB& rootBounds, uint32_t rootIndex) {
    mpDeviceNodeData =
        std::make_unique<DeviceBuffer>(sizeof(BVHNode) * mNodes.size());

    mpDeviceNodeData->copyFromHost(mNodes.data());

    mView.blas.bounds = rootBounds;
    mView.blas.rootIndex = rootIndex;

    // Create node data
    mView.blasNodes = (BVHNode*)mpDeviceNodeData->data();

    reorderMeshData(data, finalIndices);

    mBuildStats.nodeCount = mNodes.size();
}

void BVH::reorderMeshData(SceneData& data,
                          const std::vector<uint32_t>& finalIndices) {
    // Create reordered data for each mesh
    std::vector<MeshData> reorderedMeshes(data.meshes.size());

    for (size_t i = 0; i < finalIndices.size(); ++i) {
        const auto& tri = mBuildTris[finalIndices[i]];
        auto& srcMesh = data.meshes[tri.meshIndex];
        auto& dstMesh = reorderedMeshes[tri.meshIndex];

        for (int j = 0; j < 3; ++j) {
            uint32_t srcIdx = srcMesh.index[tri.triIndex * 3 + j];

            dstMesh.position.push_back(srcMesh.position[srcIdx]);
            dstMesh.normal.push_back(srcMesh.normal[srcIdx]);
            dstMesh.texCrd.push_back(srcMesh.texCrd[srcIdx]);
            dstMesh.index.push_back(dstMesh.position.size() - 1);
        }
    }

    data.meshes = std::move(reorderedMeshes);
}

BVH::~BVH() {}

}  // namespace CRay
