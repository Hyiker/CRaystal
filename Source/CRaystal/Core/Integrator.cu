#include "hip/hip_runtime.h"
#include "Core/Sampler.h"
#include "Integrator.h"
#include "Utils/Progress.h"

namespace CRay {
PathTraceIntegrator::PathTraceIntegrator() {
    mpConstDataBuffer = std::make_unique<DeviceBuffer>(sizeof(DeviceView));
}

__global__ void pathTraceKernel(uint32_t frameIdx,
                                const PathTraceIntegratorView* pIntegrator,
                                const SceneView* pScene,
                                const CameraProxy* pCamera,
                                SensorData* pSensor) {
    UInt2 xy(blockIdx.x * blockDim.x + threadIdx.x,
             blockIdx.y * blockDim.y + threadIdx.y);

    UInt2 sensorSize = pSensor->size;
    if (xy.x >= sensorSize.x || xy.y >= sensorSize.y) {
        return;
    }

    Sampler sampler(xy, frameIdx);
    Float2 pixel = Float2(xy) + sampler.nextSample2D();

    auto ray = pCamera->generateRay(sensorSize, pixel);

    Spectrum color;

    RayHit rayHit;
    rayHit.ray = ray;
    if (pScene->intersect(rayHit)) {
        const Intersection it = pScene->createIntersection(rayHit);
        uint32_t materialID =
            pScene->meshSOA.getMeshDesc(rayHit.hitInfo.primitiveIndex)
                .materialID;

        MaterialData materialData =
            pScene->materialSystem.getMaterialData(materialID);
        if (materialData.isEmissive()) {
            color = Spectrum(materialData.emission);
        } else {
            color = Spectrum(materialData.diffuseRefl);
        }
    }

    pSensor->addSample(color, pixel);
}

PathTraceIntegratorView* PathTraceIntegrator::getDeviceView() const {
    return (PathTraceIntegratorView*)mpConstDataBuffer->data();
}

void PathTraceIntegrator::dispatch(Scene& scene, int spp) const {
    mpConstDataBuffer->copyFromHost(&mView);

    auto pCamera = scene.getCamera();
    auto pSensor = pCamera->getSensor();

    pSensor->setSPP(spp);

    pSensor->updateDeviceData();
    pCamera->updateDeviceData();

    UInt2 size = pSensor->getSize();

    for (int i : Progress(pSensor->getSPP(), "Render progress ")) {
        pathTraceKernel<<<dim3(size.x, size.y, 1), dim3(16, 16, 1)>>>(
            i, getDeviceView(), scene.getDeviceView(), pCamera->getDeviceView(),
            pSensor->getDeviceView());

        hipDeviceSynchronize();
    }

    pSensor->readbackDeviceData();
}

}  // namespace CRay
