#include "hip/hip_runtime.h"

#include "CRaystal.h"
#include "Core/Sampler.h"
#include "Utils/Progress.h"
#include "Walkthrough.h"

namespace CRay {

__global__ void renderFrame(uint32_t frame, const SceneView* pScene,
                            const CameraProxy* pCamera, SensorData* pSensor) {
    UInt2 xy(blockIdx.x * blockDim.x + threadIdx.x,
             blockIdx.y * blockDim.y + threadIdx.y);

    UInt2 sensorSize = pSensor->size;
    if (xy.x >= sensorSize.x || xy.y >= sensorSize.y) {
        return;
    }

    Sampler sampler(xy, frame);
    Float2 pixel = Float2(xy) + sampler.nextSample2D();

    auto ray = pCamera->generateRay(sensorSize, pixel);
    Spectrum color;

    RayHit rayHit;
    rayHit.ray = ray;
    if (pScene->intersect(rayHit)) {
        const Intersection it = pScene->createIntersection(rayHit);
        uint32_t materialID =
            pScene->meshSOA.getMeshDesc(rayHit.hitInfo.primitiveIndex)
                .materialID;

        MaterialData materialData =
            pScene->materialSystem.getMaterialData(materialID);

        color = Spectrum(materialData.diffuseRefl);
    }

    pSensor->addSample(color, pixel);
}

void crayRenderSample(const Scene::Ref& pScene, int spp) {
    auto pCamera = pScene->getCamera();
    auto pSensor = pCamera->getSensor();

    pSensor->setSPP(spp);

    pSensor->updateDeviceData();
    pCamera->updateDeviceData();

    UInt2 size = pSensor->getSize();

    for (int i : Progress(pSensor->getSPP(), "Render progress ")) {
        renderFrame<<<dim3(size.x, size.y, 1), dim3(16, 16, 1)>>>(
            i, pScene->getDeviceView(), pCamera->getDeviceView(),
            pSensor->getDeviceView());

        hipDeviceSynchronize();
    }

    pSensor->readbackDeviceData();
    auto pImage = pSensor->createImage();
    pImage->writeEXR("walkthrough.exr");
}

}  // namespace CRay
