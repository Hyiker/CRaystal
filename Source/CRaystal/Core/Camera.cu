#include <hip/hip_runtime.h>

#include "Camera.h"
#include "Error.h"
namespace CRay {

Camera::Camera() : mpDeviceData(nullptr) {
    hipMalloc(&mpDeviceData, sizeof(CameraProxy));

    updateDeviceData();
}

Camera::Camera(Camera&& other) noexcept
    : mData(std::move(other.mData)), mpDeviceData(other.mpDeviceData) {
    other.mpDeviceData = nullptr;
}

Camera& Camera::operator=(Camera&& other) noexcept {
    if (this != &other) {
        if (mpDeviceData) {
            hipFree(mpDeviceData);
            mpDeviceData = nullptr;
        }

        mData = std::move(other.mData);

        mpDeviceData = other.mpDeviceData;
        other.mpDeviceData = nullptr;
    }
    return *this;
}

void Camera::calculateCameraData() const {
    mData.cameraW = normalize(mData.target - mData.posW);
    mData.cameraU = normalize(cross(mData.cameraW, mData.up));
    mData.cameraV = normalize(cross(mData.cameraU, mData.cameraW));
}

void Camera::updateDeviceData() const {
    CRAYSTAL_CHECK(mpDeviceData != nullptr, "Cuda pointer is none");
    hipMemcpy(mpDeviceData, &mData, sizeof(CameraProxy),
               hipMemcpyHostToDevice);
}

Camera::~Camera() {
    if (mpDeviceData) {
        hipFree(mpDeviceData);
        mpDeviceData = nullptr;
    }
}
}  // namespace CRay
