#include "hip/hip_runtime.h"
#include "MathDefs.h"
#include "Sampling.h"
namespace CRay {
CRAYSTAL_API CRAYSTAL_DEVICE_HOST Float2 sampleBarycentric(const Float2& u) {
    float u1 = u[0];
    float u2 = u[1];

    bool flip = (u1 + u2) > 1.0f;
    u1 = flip ? (1.0f - u1) : u1;
    u2 = flip ? (1.0f - u2) : u2;

    return Float2(u1, u2);
}

CRAYSTAL_DEVICE_HOST Float3 uniformSampleSphere(const Float2& u) {
    Float z = 1.0f - 2.0f * u[0];
    Float r = std::sqrt(std::max(0.0f, 1.0f - z * z));
    Float phi = 2.0f * kPi * u[1];
    Float x = r * std::cos(phi);
    Float y = r * std::sin(phi);
    return Float3(x, y, z);
}

CRAYSTAL_DEVICE_HOST Float uniformSampleSpherePdf() { return 1.0f * kInv4Pi; }

CRAYSTAL_DEVICE_HOST Float3 uniformSampleHemisphere(const Float2& u) {
    Float z = u[0];
    Float r = std::sqrt(std::max(0.0f, 1.0f - z * z));
    Float phi = 2.0f * kPi * u[1];
    Float x = r * std::cos(phi);
    Float y = r * std::sin(phi);
    return Float3(x, y, z);
}

CRAYSTAL_DEVICE_HOST Float uniformSampleHemispherePdf() {
    return 1.0f * kInv2Pi;
}

CRAYSTAL_DEVICE_HOST Float3 cosineWeightSampleHemisphere(const Float2& u) {
    // Using Malley's method
    Float r = std::sqrt(u[0]);
    Float phi = 2.0f * kPi * u[1];
    Float x = r * std::cos(phi);
    Float y = r * std::sin(phi);
    Float z = std::sqrt(std::max(0.0f, 1.0f - x * x - y * y));
    return Float3(x, y, z);
}

CRAYSTAL_DEVICE_HOST Float cosineWeightSampleHemispherePdf(const Float3& d) {
    return std::max(0.0f, d.z) * kInvPi;
}

CRAYSTAL_DEVICE_HOST Float powerHeuristic(int nf, Float fPdf, int ng,
                                          Float gPdf) {
    Float f = nf * fPdf;
    Float g = ng * gPdf;
    return (f * f) / (f * f + g * g);
}

}  // namespace CRay
